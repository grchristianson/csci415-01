#include "hip/hip_runtime.h"
//
// Assignment 1: ParallelSine
// CSCI 415: Networking and Parallel Computation
// Spring 2017
// Name(s): 
//
// Sine implementation derived from slides here: http://15418.courses.cs.cmu.edu/spring2016/lecture/basicarch


// standard imports
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>

// problem size (vector length) N
static const int N = 12345678;

// Number of terms to use when approximating sine
static const int TERMS = 6;

// kernel function (CPU - Do not modify)
void sine_serial(float *input, float *output)
{
  int i;

  for (i=0; i<N; i++) {
      float value = input[i]; 
      float numer = input[i] * input[i] * input[i]; 
      int denom = 6; // 3! 
      int sign = -1; 
      for (int j=1; j<=TERMS;j++) 
      { 
         value += sign * numer / denom; 
         numer *= input[i] * input[i]; 
         denom *= (2*j+2) * (2*j+3); 
         sign *= -1; 
      } 
      output[i] = value; 
    }
}


// kernel function (CUDA device)
// TODO: Implement your graphics kernel here. See assignment instructions for method information  


//implement kernel
__global__ void sine_parallel(float *input, float *output)
{
      //calculate array index
      int idx = blockIdx.x * blockDim.x + threadIdx.x;

      //perform taylor expansion

      float value = input[idx];
      float numer = input[idx] * input[idx] * input[idx];
      int denom = 6; // 3!
      int sign = -1;
      for (int j=1; j<=TERMS;j++)
      {
         value += sign * numer / denom;
         numer *= input[idx] * input[idx];
         denom *= (2*j+2) * (2*j+3);
         sign *= -1;
      }
 
      output[idx] = value;

}


// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

// END: timing and error checking routines (do not modify)



int main (int argc, char **argv)
{
  //BEGIN: CPU implementation (do not modify)
  float *h_cpu_result = (float*)malloc(N*sizeof(float));
  float *h_input = (float*)malloc(N*sizeof(float));
  //Initialize data on CPU
  int i;
  for (i=0; i<N; i++)
  {
    h_input[i] = 0.1f * i;
  }

  //Execute and time the CPU version
  long long CPU_start_time = start_timer();
  sine_serial(h_input, h_cpu_result);
  long long CPU_time = stop_timer(CPU_start_time, "\nCPU Run Time");
  //END: CPU implementation (do not modify)


  //TODO: Prepare and run your kernel, make sure to copy your results back into h_gpu_result and display your timing results
  float *h_gpu_result = (float*)malloc(N*sizeof(float));


  //gpu device in and out pointers
  float *d_in;
  float *d_out;

  //GPU Memory Allocation
  long long GPU_Mem_Alloc_start_time = start_timer();
  hipMalloc((void **) &d_in, (N*sizeof(float)));
  hipMalloc((void **) &d_out, (N*sizeof(float)));
  long long GPU_Mem_Alloc_time = stop_timer(GPU_Mem_Alloc_start_time, "\nGPU Memory Allocation: ");

  //GPU Memory Copy to Device
  long long GPU_Mem_copy_start_time = start_timer();
  hipMemcpy(d_in, h_input, (N*sizeof(float)), hipMemcpyHostToDevice);
  long long GPU_Memcpy_time = stop_timer(GPU_Mem_copy_start_time, "\nGPU Memory Copy to Device: ");


  //GPU Kernel Run (1024 maximum threads 12345678/1024 rounded blocks =12057)

  long long GPU_execute_start_time = start_timer();
  sine_parallel<<<12057, 1024>>>(d_in, d_out);
  long long GPU_execute_time = stop_timer(GPU_execute_start_time, "\nGPU Kernel Run Time");
  

  //GPU Memory Copy to Host
  long long GPU_host_start_time = start_timer();
  hipMemcpy(h_gpu_result,d_out, N*sizeof(float), hipMemcpyDeviceToHost);
  long long GPU_host_time = stop_timer(GPU_host_start_time, "\nGPU Copy Memory to Host Time");
  


  // Checking to make sure the CPU and GPU results match - Do not modify
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");

  // Cleaning up memory
  free(h_input);
  free(h_cpu_result);
  free(h_gpu_result);
  return 0;
}






